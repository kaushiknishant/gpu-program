#include <iostream>
#include <hip/hip_runtime.h>


__host__ __device__ float f(float a, float b){
    return a + b;
}

void vecadd_cpu(float* x, float* y, float* z, int N){
    for(unsigned int i = 0; i < N; ++i){
        z[i] = f(x[i], y[i]);
    }
}

__global__ void vecadd_kernel(float* x, float* y, float* z, int N){
    // Grid Dimension ---> gridDim.x tells no. of blocks in the grid
    // Block Index ---> blockIdx.x  tells index of its block w.r.t other blocks in grid
    // Block dimension ---> blockDim.x tells threads in block(size of block)
    // Thread Index ---> threadIdx.x tells position of thread in block
    unsigned int i = blockDim.x*blockIdx.x+threadIdx.x;
    if(i < N){
        z[i] = f(x[i], y[i]);
    }

    // single program multiple data 

}

void vecadd_gpu(float* x, float* y, float* z, int N){
    // Allocate GPU memory
    float *x_d, *y_d, *z_d;
    hipMalloc((void**)&x_d, N*sizeof(float));
    hipMalloc((void**)&y_d, N*sizeof(float));
    hipMalloc((void**)&z_d, N*sizeof(float));

    // Copy to the GPU
    hipMemcpy(x_d, x, N*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N*sizeof(float),hipMemcpyHostToDevice);

    // Run the GPU code
    // call a GPU kernel function(launch a grid of threads)
    const unsigned int numThreadsPerBlock = 512;
    const unsigned int numBlocks = (N + numThreadsPerBlock - 1)/512;
    vecadd_kernel<<< numBlocks, numThreadsPerBlock >>>(x_d, y_d, z_d, N);

    // Copy from the GPU
    hipMemcpy(z, z_d, N*sizeof(float),hipMemcpyDeviceToHost);

    // Deallocate GPu memory
    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}


int main(int argc, char**argv) {

    hipDeviceSynchronize();
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    // Allocate memory and initialize data
    unsigned int N = (argc > 1)?(atoi(argv[1])): (1 << 25) ;
    float* x = (float*) malloc(N*sizeof(float));
    float* y = (float*) malloc(N*sizeof(float));
    float* z = (float*) malloc(N*sizeof(float));
    for (unsigned int i = 0; i < N; ++i) {
        x[i] = rand();
        y[i] = rand();
    }

    //vector addition on CPU
    hipEventRecord(start);
    vecadd_cpu(x,y,z,N);
    hipEventRecord(end);
    hipEventSynchronize(end);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "CPU program took " << milliseconds << " milliseconds to execute." << std::endl;

    //vector addition on GPU
    hipEventRecord(start);
    vecadd_gpu(x,y,z,N);
    hipEventRecord(end);
    hipEventSynchronize(end);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, end);
    std::cout << "GPU program took " << milliseconds << " milliseconds to execute." << std::endl;
    
    hipEventDestroy(start);
    hipEventDestroy(end);
}